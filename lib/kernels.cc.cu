#include "hip/hip_runtime.h"
// This file contains the GPU implementation of our op. It's a pretty typical CUDA kernel
// and I make no promises about the quality of the code or the choices made therein, but
// it should get the point accross.

#include "ehrlich_aberth.h"
#include "kernel_helpers.h"
#include "kernels.h"

namespace ehrlich_aberth_jax {

namespace {

__global__ void ehrlich_aberth_kernel(std::int64_t size, std::int64_t deg,
                                      const thrust::complex<double> *poly,
                                      thrust::complex<double> *roots) {
  const std::int64_t itmax = 50;

  // Compute roots
  std::int64_t i;
  for (std::int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += blockDim.x * gridDim.x) {
    i = idx * (deg + 1);
    ehrlich_aberth(poly + i, roots + i - idx, deg, itmax);
  }
}

void ThrowIfError(hipError_t error) {
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

inline void apply_ehrlich_aberth(hipStream_t stream, void **buffers, const char *opaque,
                                 std::size_t opaque_len) {
  const EhrlichAberthDescriptor &d =
      *UnpackDescriptor<EhrlichAberthDescriptor>(opaque, opaque_len);
  const std::int64_t size = d.size;
  const std::int64_t deg = d.deg;

  const thrust::complex<double> *poly =
      reinterpret_cast<const thrust::complex<double> *>(buffers[0]);
  thrust::complex<double> *roots = reinterpret_cast<thrust::complex<double> *>(buffers[1]);

  const int block_dim = 128;
  const int grid_dim = std::min<int>(1024, (size + block_dim - 1) / block_dim);
  ehrlich_aberth_kernel<<<grid_dim, block_dim, 0, stream>>>(size, deg, poly, roots);

  ThrowIfError(hipGetLastError());
}

}  // namespace

void gpu_ehrlich_aberth(hipStream_t stream, void **buffers, const char *opaque,
                        std::size_t opaque_len) {
  apply_ehrlich_aberth(stream, buffers, opaque, opaque_len);
}

}  // namespace ehrlich_aberth_jax
